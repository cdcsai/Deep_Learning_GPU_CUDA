#include "hip/hip_runtime.h"

#include "device_atomic_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

using namespace std;

__global__ void dot(int *aa, int *bb, int *cc)
{
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = aa[index] * bb[index];

    __syncthreads();

    if (0 == threadIdx.x)
    {
        int sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }
        atomicAdd(cc, sum);
    }
}

int main()
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c(0);
    int size = N * sizeof(int);

   //allocate space for the variables on the device
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, sizeof(int));

   //allocate space for the variables on the host
   a = (int *)malloc(size);
   b = (int *)malloc(size);
   c = (int *)malloc(sizeof(int));

   //this is our ground truth
   int sumTest = 0;
   //generate numbers
   for (int i = 0; i < N; i++)
   {
       //a[i] = rand() % 10;
       //b[i] = rand() % 10;
	   a[i] = i;
	   b[i] = 10;
       sumTest += a[i] * b[i];
       //printf(" %d %d \n",a[i],b[i]);
   }

   //*c = 0;

   hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
   hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);

   dot<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c);

   hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
   cout << "c: " << *c << endl;
   cout << "sumTest: " << sumTest << endl;
   //printf("%i ", *c);
   //printf("%d ", sumTest);

   free(a);
   free(b);
   free(c);

   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   return 0;
 }
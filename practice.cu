
#include <hip/hip_runtime.h>
# include <stdio.h>


__global__ void cube(float *d_in, float *d_out){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f * f;
}

int main(){
  float *h_in, *h_out;
  const int ARRAYSIZE = 64;
  const int ARRAYBYTE = 64 * sizeof(float);

  // Generate input ARRAY

  h_in = (float*)malloc(ARRAYBYTE);
	h_out = (float*)malloc(ARRAYBYTE);

  for (int i = 0; i<ARRAYSIZE; i++){
    h_in[i] = float(i);
  }


  // Declare GPU memory pointer

  float * d_in;
  float * d_out;

  hipMalloc((void **) &d_in, ARRAYBYTE);
  hipMalloc((void **) &d_out, ARRAYBYTE);

  hipMemcpy(d_in, h_in, ARRAYBYTE, hipMemcpyHostToDevice);
  cube<<<1, ARRAYSIZE>>>(d_out, d_in);
  hipMemcpy(h_out, d_out, ARRAYBYTE, hipMemcpyDeviceToHost);

  for (int i = 0; i<ARRAYSIZE; i++){
    printf("%f\n", h_out[i]);
  free(h_in);
  free(h_out);
  hipFree(d_in);
  hipFree(d_out);

  return 0;
  }
}
